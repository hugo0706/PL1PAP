#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdlib.h>

#define WIDTH 16

__global__ void convolucion(int* c_d, int* a_d, int* b_d)
{

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int col = (bx * ((WIDTH / 2) - 1) + tx);
    int row = (by * ((WIDTH / 2) - 1) + ty);

    if (tx < (WIDTH / 2) && tx > 0 && ty < (WIDTH / 2) && ty > 0) {
        for (int dcol = -1; dcol <= 1; dcol++) {
            for (int drow = -1; drow <= 1; drow++) {
                c_d[(col - 1) + ((row - 1) * ((WIDTH)-2))] += a_d[(col + dcol) + (row + drow) *(WIDTH/2)]* b_d[(dcol + 1) + (drow + 1) * 3];
            }
        }
    }
}

void printMatrix(const int* m, const int rows, const int cols)
{
    for (int i = 0; i < rows; i++)
    {
        printf("%s{", (i == 0 ? "{" : " "));
        for (int j = 0; j < cols; j++)
            printf("%s%5d", (j == 0 ? "" : ","), m[j + i * cols]);
        printf("}%s\n", (i == rows - 1 ? "}" : ""));
    }
}

int main()
{
    //declaramos 
    int* a_h, * b_h, * c_h, * a_d, * b_d, * c_d;

    //reservar mem CPU 
    a_h = (int*)malloc((WIDTH * WIDTH) * sizeof(int));
    b_h = (int*)malloc(9 * sizeof(int));
    c_h = (int*)malloc(((WIDTH - 2) * (WIDTH - 2)) * sizeof(int));

    //reservar mem GPU 
    hipMalloc(&a_d, (WIDTH * WIDTH) * sizeof(int));
    hipMalloc(&b_d, 9 * sizeof(int));
    hipMalloc(&c_d, ((WIDTH - 2) * (WIDTH - 2)) * sizeof(int));

    srand(time(NULL));

    for (int i = 0; i < (WIDTH * WIDTH); i++) {
        a_h[i] =  rand() % 256;
    }

    for (int i = 0; i < 9; i++) {
        b_h[i] = rand() % 10;
    }

    hipMemcpy(a_d, a_h, (WIDTH * WIDTH) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, 9 * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(((WIDTH / 2) + 1), ((WIDTH / 2) + 1));
    dim3 gridDim(2, 2);

    convolucion << < gridDim, blockDim >> > (c_d, a_d, b_d);
    hipMemcpy(c_h, c_d, ((WIDTH - 2) * (WIDTH - 2)) * sizeof(int), hipMemcpyDeviceToHost);


    printf("Matriz A:\n");
    printMatrix(a_h, 16, 16);
    printf("Matriz B:\n");
    printMatrix(b_h, 3, 3);
    printf("Matriz C:\n");
    printMatrix(c_h, 14, 14);

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    free(a_h);
    free(b_h);
    free(c_h);

    return 0;
}